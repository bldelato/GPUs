#include <stdio.h>
#include "hipblas.h"
#include "matrix_mul.h"
#include "hipblas.h"

#define DEBUG 0

// Host multiplication function
// Compute C = A * B
// hA is the height of A
// wA is the width of A
// wB is the width of B

extern "C"
void Mul(float* A, float* B, int hA, int wA, int wB, float* C) {
	int size;
	const float alpha = 1.0f;
	const float beta = 0.0f;

	#if (DEBUG > 0)
	printf("Checkpoint! Begin Mul call (inside)\n");
	#endif

	// Load A and B to the device
	float* Ad;
	size = hA * wA * sizeof(float);
	hipMalloc((void**)&Ad, size);
	hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
	float* Bd;
	size = wA * wB * sizeof(float);
	hipMalloc((void**)&Bd, size);
	hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);

	// Allocate C on the device
	float* Cd;
	size = hA * wB * sizeof(float);
	hipMalloc((void**)&Cd, size);

	#if (DEBUG > 0)
	printf("Checkpoint! Middle Mul call (inside) Memory allocated\n");
	#endif

	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasStatus_t ret;

	#if (DEBUG > 0)
	printf("Checkpoint! Middle Mul call (inside) About to call hipblasSgemm\n");
	#endif

	// Compute the execution configuration
	ret = hipblasSgemm(
		handle,				/*hipblasHandle_t handle */
		/* hipblasOperation_t: HIPBLAS_OP_N the non-transpose operation is selected | HIPBLAS_OP_T the transpose operation is selected | HIPBLAS_OP_C the conjugate transpose operation is selected */
		HIPBLAS_OP_N,			/* hipblasOperation_t transa */
		HIPBLAS_OP_N,			/* hipblasOperation_t transb */
		hA,				/* [m] */ 
		wB,				/* [n] */  
		wA,				/* [k] */ 
		&alpha,				/* alfa */ 
		Bd, wB,				/* A[m][k], num columnas (lda) */ 
		Ad, wA,				/* B[k][n], num columnas (ldb) */
		&beta,				/* beta */
		Cd, wA				/* C[m][n], num columnas (ldc) */
	);
	#if (DEBUG > 0)
	printf("Checkpoint! Middle Mul call (inside) Called hipblasSgemm\n");
	#endif
	if (ret != HIPBLAS_STATUS_SUCCESS) {
		printf("hipblasSgemm returned error code %d, line(%d)\n", ret, __LINE__);
	}

	// Read C from the device
	hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(Ad);
	hipFree(Bd);
	hipFree(Cd);

	#if (DEBUG > 0)
	printf("Checkpoint! End Mul call (inside)\n");
	#endif
}
